#include "int_array.cuh"

#include "cnpy.h"

namespace {
    using namespace thesis;
    constexpr auto type_size = sizeof(decltype(HostIntArray::i)::value_type);
};

thesis::HostIntArray::HostIntArray(const size_t count) : i(count) {}

void thesis::HostIntArray::copy(const DeviceIntArray &indices) {
    const auto size = i.size() * type_size;
    hipMemcpy(i.data(), indices.i, size, hipMemcpyDeviceToHost);
}

void thesis::HostIntArray::save(std::filesystem::path filepath) {
    const auto shape = std::vector{ i.size() };
    cnpy::npy_save(filepath, i.data(), shape);
}

thesis::DeviceIntArray::DeviceIntArray(const HostIntArray &indices) {
    const auto size = indices.i.size() * type_size;
    hipMalloc(&i, size);
}

thesis::DeviceIntArray::~DeviceIntArray() {
    hipFree(i);
}

void thesis::DeviceIntArray::copy(const HostIntArray &indices) {
    const auto size = indices.i.size() * type_size;
    hipMemcpy(i, indices.i.data(), size, hipMemcpyHostToDevice);
}
