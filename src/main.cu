#include "hip/hip_runtime.h"
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#include <cub/device/device_radix_sort.cuh>

#include "grid.cuh"
#include "int_array.cuh"
#include "particles.cuh"

//#define DEBUG

enum class Version {
    global = 0,
    shared
};

// XXX: Hardcoded block_size.
#ifndef DEBUG
const auto block_size = 128;
#else
const auto block_size = 1;
#endif

auto generate_particles_from_2d_pattern(
    const int3 simulation_dimensions, const int cell_size,
    const int particles_per_cell, const float particle_charge,
    const int random_seed
) {
    /* 
     * The simulation box will be split into four zones that determine the
     * particle density from low to high density, all relative to the
     * user-specified number of particles per cell:
     * ┌────────────────────────────────┐
     * │ 3.            mid              │
     * ├────────┬───────────────────────┤
     * │ 1. low │ 2.   high->low        │
     * ├────────┴───────────────────────┤
     * │ 0.            mid              │
     * └────────────────────────────────┘
     * The following diagram shows the size of each zone using equally sized
     * boxes:
     * ┌───┬───┬───┬───┐
     * │ 3 │ 3 │ 3 │ 3 │
     * ├───┼───┼───┼───┤
     * │ 1 │ 2 │ 2 │ 2 │
     * ├───┼───┼───┼───┤
     * │ 1 │ 2 │ 2 │ 2 │
     * ├───┼───┼───┼───┤
     * │ 0 │ 0 │ 0 │ 0 │
     * └───┴───┴───┴───┘
     */
    auto random_engine = std::default_random_engine(random_seed);
    // Particle density distributions.
    auto low_density_distribution = std::uniform_int_distribution<int>(0, 4);
    auto mid_density_distribution = std::uniform_int_distribution<int>(
        0.5 * particles_per_cell, 1.5 * particles_per_cell
    );
    auto high_density_distribution = std::uniform_int_distribution<int>(
        1.5 * particles_per_cell, 2 * particles_per_cell
    );

    // Shorthand notations.
    const auto I = simulation_dimensions.x;
    const auto J = simulation_dimensions.y;

    auto particle_densities = std::vector<int>(I * J);
    auto particle_count = 0;
    
    // Mid density distribution (zone 0 and 3).
    for (auto j = 0; j < J; ++j) {
        for (auto i = 0; i < I; ++i) {
            const auto cell_index = i + j * I;
            // Skip zone 1 and 2.
            if (j >= J/4 && j < J * 3/4) {
                continue;
            }
            const auto cell_particle_count = mid_density_distribution(
                random_engine
            );
            particle_densities[cell_index] = cell_particle_count;
            particle_count += cell_particle_count;
        }
    }
    // Low distribution (zone 1).
    for (auto j = J / 4; j < J * 3/4; ++j) {
        for (auto i = 0; i < I / 4; ++i) {
            const auto cell_index = i + j * I;
            const auto cell_particle_count = low_density_distribution(
                random_engine
            );
            particle_densities[cell_index] = cell_particle_count;
            particle_count += cell_particle_count;
        }
    }
    // Linear gradient distribution (zone 2).
    for (auto j = J / 4; j < J * 3/4; ++j) {
        for (auto i = I / 4; i < I; ++i) {
            const auto cell_index = i + j * I;
            
            const auto mid_density = mid_density_distribution(random_engine);
            const auto high_density = high_density_distribution(random_engine);
            // Linear gradient from high (left) to low (right).
            const auto zone_start = I / 4;
            const auto zone_width = I * 3.0f/4.0f;
            const auto t = (i - zone_start) / zone_width;
            const auto cell_particle_count = static_cast<int>(
                t * mid_density + (1 - t) * high_density
            );

            particle_densities[cell_index] = cell_particle_count;
            particle_count += cell_particle_count;
        }
    }

    // Generate particles from particle densities.
    auto position_distribution = std::uniform_real_distribution<float>(
        0, cell_size
    );
    auto particles = amitis::HostParticles{ particle_count, particle_charge };
    auto particle_index = 0;
    for (auto j = 0; j < J; ++j) {
        const auto y_offset = j * cell_size;
        for (auto i = 0; i < I; ++i) {
            const auto x_offset = i * cell_size;
            const auto cell_index = i + j * I;
            const auto cell_particle_count = particle_densities[cell_index];
            for (auto p = 0; p < cell_particle_count; ++p) {
                particles.pos_x[particle_index] = x_offset
                    + position_distribution(random_engine);
                particles.pos_y[particle_index] = y_offset
                    + position_distribution(random_engine);
                ++particle_index;
            }
        }
    }

    return particles;
}

auto generate_particles(
    const int3 simulation_dimensions, const int cell_size,
    const int particles_per_cell, const float particle_charge,
    const int random_seed
) -> amitis::HostParticles {
    auto random_engine = std::default_random_engine(random_seed);
    auto distribution_x = std::uniform_real_distribution<float>(
        0, cell_size
    );
    auto distribution_y = std::uniform_real_distribution<float>(
        0, cell_size
    );
    auto distribution_z = std::uniform_real_distribution<float>(
        0, cell_size
    );

    const auto particle_count = particles_per_cell * simulation_dimensions.x
                                                   * simulation_dimensions.y
                                                   * simulation_dimensions.z;
    auto particles = amitis::HostParticles{ particle_count, particle_charge };

    auto particle_index = 0;
    for (auto k = 0; k < simulation_dimensions.z; ++k) {
        const auto z_offset = k * cell_size;
        for (auto j = 0; j < simulation_dimensions.y; ++j) {
            const auto y_offset = j * cell_size;
            for (auto i = 0; i < simulation_dimensions.x; ++i) {
                const auto x_offset = i * cell_size;
                for (auto p = 0; p < particles_per_cell; ++p) {
                    particles.pos_x[particle_index] = x_offset
                        + distribution_x(random_engine);
                    particles.pos_y[particle_index] = y_offset
                        + distribution_y(random_engine);
                    particles.pos_z[particle_index] = z_offset
                        + distribution_z(random_engine);
                    ++particle_index;
                }
            }
        }
    }

    return particles;
}

constexpr auto cell_coordinates(const float3 position, const int cell_size) {
    // XXX: Hardcoded half-cell shift due to one layer of ghost cells.
    return float3{
        position.x / cell_size + 0.5f,
        position.y / cell_size + 0.5f,
        position.z / cell_size + 0.5f
    };
}

constexpr auto cell_index(const int3 cell_center,
        const int3 grid_dimensions) {
    const auto i = cell_center.x;
    const auto j = cell_center.y;
    const auto k = cell_center.z;
    return i + (j * grid_dimensions.x)
             + (k * grid_dimensions.x * grid_dimensions.y);
}

/// https://graphics.stanford.edu/%7Eseander/bithacks.html#RoundUpPowerOf2
constexpr
auto ceil_pow2(const int number) -> int {
    auto v = static_cast<uint32_t>(number);
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    v += v == 0;
    return static_cast<int>(v);
}

__global__
void initialize_indices(int *indices, const size_t particle_count) {
    // Grid-stride loop. Equivalent to regular if-statement grid is large enough
    // to cover all iterations of the loop.
    for (
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        index < particle_count;
        index += blockDim.x * gridDim.x
    ) {
    indices[index] = index;
    }
}

/// Compute the index of the first (by index) enclosing cell for each particle.
__global__
void initialize_particle_cell_indices(
    const float *pos_x, const float *pos_y, const size_t particle_count,
    int *cell_indices, const int3 grid_dimensions, const int cell_size
) {
    // Grid-stride loop. Equivalent to regular if-statement grid is large enough
    // to cover all iterations of the loop.
    for (
        auto particle_index = blockIdx.x * blockDim.x + threadIdx.x;
        particle_index < particle_count;
        particle_index += blockDim.x * gridDim.x
    ) {
        // Position in world coordinates.
        const auto position = float3{
            pos_x[particle_index], pos_y[particle_index], 0
        };
        // Position in grid coordinates.
        const auto [ u, v, w ] = cell_coordinates(position, cell_size);
        // 2D indices of first enclosing cell, by first meaning the one with
        // lowest index, i.e., closest to the origin.
        const auto i = static_cast<int>(floor(u));
        const auto j = static_cast<int>(floor(v));

        cell_indices[particle_index] = i + j * grid_dimensions.x;
    }
}

__global__
void initialize_kernel_data(
    const size_t particle_count, const int *cell_indices,
    int *particle_indices_rel_cell, int *particle_count_per_cell
) {

}

__global__
void charge_density_global_2d(
    const float *pos_x, const float *pos_y, const size_t particle_count,
    float particle_charge, float *densities, const int3 grid_dimensions,
    const int cell_size
) {
    // Grid-stride loop. Equivalent to regular if-statement grid is large enough
    // to cover all iterations of the loop.
    for (
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        index < particle_count;
        index += blockDim.x * gridDim.x
    ) {
        const auto position = float3{ pos_x[index], pos_y[index], 0 };
        const auto [ u, v, w ] = cell_coordinates(position, cell_size);

        // 2D index, or center of surrounding cell closest to the origin.
        const auto i = static_cast<int>(floor(u));
        const auto j = static_cast<int>(floor(v));

        // Centers of all surrounding cells, named relative the indices (i,j,k)
        // of the surrounding cell closest to the origin (cell_000).
        const auto cell_000_center = int3{ i,     j    , 0 };
        const auto cell_100_center = int3{ i + 1, j    , 0 };
        const auto cell_010_center = int3{ i,     j + 1, 0 };
        const auto cell_110_center = int3{ i + 1, j + 1, 0 };

        // uvw-position relative to cell_000.
        const auto pos_rel_cell = float3{
            u - cell_000_center.x,
            v - cell_000_center.y,
            w - cell_000_center.z
        };
        // Cell weights based on the distance to the particle.
        const auto cell_000_weight = (1 - pos_rel_cell.x) * (1 - pos_rel_cell.y);
        const auto cell_100_weight =      pos_rel_cell.x  * (1 - pos_rel_cell.y);
        const auto cell_010_weight = (1 - pos_rel_cell.x) *      pos_rel_cell.y;
        const auto cell_110_weight =      pos_rel_cell.x  *      pos_rel_cell.y;

        // Linear cell indices.
        const auto cell_000_index = cell_index(cell_000_center, grid_dimensions);
        const auto cell_100_index = cell_index(cell_100_center, grid_dimensions);
        const auto cell_010_index = cell_index(cell_010_center, grid_dimensions);
        const auto cell_110_index = cell_index(cell_110_center, grid_dimensions);

        // Weighted sum of the particle's charge.
        atomicAdd(&densities[cell_000_index], particle_charge * cell_000_weight);
        atomicAdd(&densities[cell_100_index], particle_charge * cell_100_weight);
        atomicAdd(&densities[cell_010_index], particle_charge * cell_010_weight);
        atomicAdd(&densities[cell_110_index], particle_charge * cell_110_weight);
    }
}

__global__
void charge_density_shared_2d(
    const float *pos_x, const float *pos_y, const size_t particle_count,
    float particle_charge, float *densities, const int3 grid_dimensions,
    const int cell_size, int *particle_indices, int * particle_cell_indices,
    int *particle_indices_rel_cell, int *particle_count_per_cell
) {
    // Grid-stride loop. Equivalent to regular if-statement grid is large enough
    // to cover all iterations of the loop.
    for (
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        index < particle_count;
        index += blockDim.x * gridDim.x
    ) {
        // Each particle will contribute to its 4 surrounding cells.
        __shared__ float s_densities[4][block_size];

        // 1D index of first enclosing cell, i.e., with lowest index.
        const auto first_cell_index = particle_cell_indices[index];
        const auto particle_index = particle_indices[index];
        const auto particle_index_rel_cell = particle_indices_rel_cell[index];
        const auto cell_particle_count = particle_count_per_cell[index];

        // Convert 1D index to 2D.
        const auto i = first_cell_index % grid_dimensions.x;
        const auto j = first_cell_index / grid_dimensions.x;

        // Centers of all surrounding cells, named relative the indices (i,j,k)
        // of the surrounding cell closest to the origin (cell_000).
        const auto cell_000_center = int3{ i,     j    , 0 };
        const auto cell_100_center = int3{ i + 1, j    , 0 };
        const auto cell_010_center = int3{ i,     j + 1, 0 };
        const auto cell_110_center = int3{ i + 1, j + 1, 0 };

        const auto position = float3{
            pos_x[particle_index], pos_y[particle_index], 0
        };
        const auto [ u, v, w ] = cell_coordinates(position, cell_size);
        // uvw-position relative to cell_000.
        const auto pos_rel_cell = float3{
            u - cell_000_center.x,
            v - cell_000_center.y,
            w - cell_000_center.z
        };
        // Cell weights based on the distance to the particle.
        const auto cell_000_weight = (1 - pos_rel_cell.x) * (1 - pos_rel_cell.y);
        const auto cell_100_weight =      pos_rel_cell.x  * (1 - pos_rel_cell.y);
        const auto cell_010_weight = (1 - pos_rel_cell.x) *      pos_rel_cell.y;
        const auto cell_110_weight =      pos_rel_cell.x  *      pos_rel_cell.y;

        // Linear cell indices.
        const auto cell_000_index = cell_index(cell_000_center, grid_dimensions);
        const auto cell_100_index = cell_index(cell_100_center, grid_dimensions);
        const auto cell_010_index = cell_index(cell_010_center, grid_dimensions);
        const auto cell_110_index = cell_index(cell_110_center, grid_dimensions);

        // Weighted sum of the particle's charge.
        s_densities[0][threadIdx.x] = particle_charge * cell_000_weight;
        s_densities[1][threadIdx.x] = particle_charge * cell_100_weight;
        s_densities[2][threadIdx.x] = particle_charge * cell_010_weight;
        s_densities[3][threadIdx.x] = particle_charge * cell_110_weight;
        // Wait until the shared memory is filled.
        __syncthreads();

        // In-place reduction in shared memory.
        for (
            auto stride = ceil_pow2(cell_particle_count) / 2;
            stride > 0;
            stride /= 2
        ) {
            // Shorthand notation.
            const auto i = particle_index_rel_cell;
            // Make sure not to stride outside of the cell range. Crucial when
            // the number of particles in a cell isn't a power of two.
            if (i < stride && i + stride < cell_particle_count) {
                s_densities[0][threadIdx.x] += s_densities[0][threadIdx.x + stride];
                s_densities[1][threadIdx.x] += s_densities[1][threadIdx.x + stride];
                s_densities[2][threadIdx.x] += s_densities[2][threadIdx.x + stride];
                s_densities[3][threadIdx.x] += s_densities[3][threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Store reduction to global memory.
        if (particle_index_rel_cell == 0) {
            atomicAdd(&densities[cell_000_index], s_densities[0][threadIdx.x]);
            atomicAdd(&densities[cell_100_index], s_densities[1][threadIdx.x]);
            atomicAdd(&densities[cell_010_index], s_densities[2][threadIdx.x]);
            atomicAdd(&densities[cell_110_index], s_densities[3][threadIdx.x]);
        }
    }
}

int main(int argc, char *argv[]) {
    using namespace amitis;

    // Unit charge.
    const auto particle_charge = 1.0f;
    // Number of outside layers of ghost cells.
    const auto ghost_layer_count = 1;

    if (argc < 8) {
        std::cerr << "Usage: "<< argv[0] << " dim_x dim_y dim_z cell_size"
            " particles/cell version output_directory [seed]\n";
        return 1;
    }

    const auto simulation_dimensions = int3{
        std::stoi(argv[1]),
        std::stoi(argv[2]),
        std::stoi(argv[3])
    };
    const auto cell_size = std::stoi(argv[4]);
    const auto particles_per_cell = std::stoi(argv[5]);
    const auto selected_version = Version{ std::stoi(argv[6]) };
    const auto output_directory_name = argv[7];
    const auto random_seed = argc > 8 ? std::stoi(argv[8]) : 1;

    // The complete grid includes ghost layers around the simulation grid.
    const auto grid_dimensions = int3{
        simulation_dimensions.x + 2 * ghost_layer_count,
        simulation_dimensions.y + 2 * ghost_layer_count,
        simulation_dimensions.z + 2 * ghost_layer_count
    };

    // Initialize particles.
    /* auto h_particles = generate_particles( */
    auto h_particles = generate_particles_from_2d_pattern(
        simulation_dimensions, cell_size, particles_per_cell, particle_charge,
        random_seed
    );
    auto d_particles = DeviceParticles{ h_particles };
    d_particles.copy(h_particles);
    std::cout << h_particles.pos_x.size() << " particles generated.\n";
    const auto particle_count = h_particles.pos_x.size();

    // Initialize grid.
    auto h_charge_densities = HostGrid{ grid_dimensions };
    auto d_charge_densities = DeviceGrid{ grid_dimensions };

    // Kernel block settings.
#ifndef DEBUG
    const auto block_count = (particle_count + block_size - 1) / block_size;
#else
    const auto block_count = 1;
#endif

/* -------------------------------------------------------------------------- */
    auto h_particle_indices_before = HostIntArray{ particle_count };
    auto h_particle_indices_after = HostIntArray{ particle_count };
    auto d_particle_indices_before = DeviceIntArray{ h_particle_indices_before };
    auto d_particle_indices_after = DeviceIntArray{ h_particle_indices_after };

    auto h_cell_indices_before = HostIntArray{ particle_count };
    auto h_cell_indices_after = HostIntArray{ particle_count };
    auto d_cell_indices_before = DeviceIntArray{ h_cell_indices_before };
    auto d_cell_indices_after = DeviceIntArray{ h_cell_indices_after };

    auto h_particle_indices_rel_cell = HostIntArray{ particle_count };
    auto h_particle_count_per_cell = HostIntArray{ particle_count };
    auto d_particle_indices_rel_cell = DeviceIntArray{ h_particle_indices_rel_cell };
    auto d_particle_count_per_cell = DeviceIntArray{ h_particle_count_per_cell };
/* Initialize radix sort ---------------------------------------------------- */
    // Initialize radix sort.
    // Temporary storage used by radix sort.
    void *d_sort_storage = nullptr;
    auto sort_storage_byte_count = size_t{};
    // Run the sorting with uninitialized temporary storage to compute the
    // required temporary storage size.
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage, sort_storage_byte_count,
        d_cell_indices_before.i, d_cell_indices_after.i,
        d_particle_indices_before.i, d_particle_indices_after.i,
        particle_count
    );
    // Allocate temporary storage.
    hipMalloc(&d_sort_storage, sort_storage_byte_count);
/* Initialize particle indices ---------------------------------------------- */
    initialize_indices<<<block_count, block_size>>>(
        d_particle_indices_before.i, particle_count
    );
/* Get cell index per particle ---------------------------------------------- */
    // Initialize particle cell indices.
    initialize_particle_cell_indices<<<block_count, block_size>>>(
        d_particles.pos_x, d_particles.pos_y, particle_count,
        d_cell_indices_before.i, grid_dimensions, cell_size
    );

/* // Sort particle indices by cell. ---------------------------------------- */
    // Sort particle indices by cell.
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage, sort_storage_byte_count,
        d_cell_indices_before.i, d_cell_indices_after.i,
        d_particle_indices_before.i, d_particle_indices_after.i,
        particle_count
    );
/* Initialize kernel data --------------------------------------------------- */
    // Initialize kernel data.
    initialize_kernel_data<<<block_count, block_size>>>(
        particle_count, d_cell_indices_after.i, d_particle_indices_rel_cell.i,
        d_particle_count_per_cell.i
    );
/* -------------------------------------------------------------------------- */

    // Run kernel.
    switch (selected_version) {
    case Version::global:
        charge_density_global_2d<<<block_count, block_size>>>(
            d_particles.pos_x, d_particles.pos_y, particle_count, particle_charge,
            d_charge_densities.cells, grid_dimensions, cell_size
        );
        break;
    case Version::shared:
        charge_density_shared_2d<<<block_count, block_size>>>(
            d_particles.pos_x, d_particles.pos_y, particle_count, particle_charge,
            d_charge_densities.cells, grid_dimensions, cell_size,
            d_particle_indices_after.i, d_cell_indices_after.i,
            d_particle_indices_rel_cell.i, d_particle_count_per_cell.i
        );
        break;
    
    default:
        std::cerr << "Unsupported version number.\n";
        return 1;
    }

    // Copy data from the device to the host.
    h_particles.copy(d_particles);
    h_charge_densities.copy(d_charge_densities);

    // Save data to disk.
    const auto output_directory = std::filesystem::path(output_directory_name);
    std::filesystem::create_directory(output_directory);
    h_particles.save_positions(output_directory / "positions.csv");
    const auto densities_filename = ([selected_version](){
        auto filename = std::string("charge_densities");
        switch (selected_version) {
        case Version::global:
            filename += "_global";
            break;
        case Version::shared:
            filename += "_shared";
            break;
        }
        filename += ".csv";
        return filename;
    })();
    h_charge_densities.save(output_directory / densities_filename);
}
