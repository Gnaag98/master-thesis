#include "hip/hip_runtime.h"
#include "grid.cuh"

#include "cnpy.h"

thesis::HostGrid::HostGrid(const int3 dimensions)
    : cells(dimensions.x * dimensions.y * dimensions.z),
      dimensions{ dimensions } {}

void thesis::HostGrid::copy(const DeviceGrid &grid) {
    const auto size = cells.size() * sizeof(float);
    hipMemcpy(cells.data(), grid.cells, size, hipMemcpyDeviceToHost);
}

void thesis::HostGrid::save(std::filesystem::path filepath) {
    const auto shape = std::vector{ cells.size() };
    cnpy::npy_save(filepath, cells.data(), shape);
}

thesis::DeviceGrid::DeviceGrid(const int3 dimensions)
    : dimensions{ dimensions } {
    const auto cell_count = dimensions.x * dimensions.y * dimensions.z;
    hipMalloc(&cells, cell_count * sizeof(float));
}

thesis::DeviceGrid::~DeviceGrid() {
    hipFree(cells);
}

void thesis::DeviceGrid::copy(const HostGrid &grid) {
    const auto size = grid.cells.size() * sizeof(float);
    hipMemcpy(cells, grid.cells.data(), size, hipMemcpyHostToDevice);
}
