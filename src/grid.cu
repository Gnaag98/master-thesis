#include "hip/hip_runtime.h"
#include "grid.cuh"

#include "cnpy.h"

thesis::HostGrid::HostGrid(const int3 dimensions)
    : cells(dimensions.x * dimensions.y * dimensions.z),
      dimensions{ dimensions } {}

void thesis::HostGrid::copy(const DeviceGrid &grid) {
    const auto size = cells.size() * sizeof(FP);
    hipMemcpy(cells.data(), grid.cells, size, hipMemcpyDeviceToHost);
}

void thesis::HostGrid::save(std::filesystem::path filepath) {
    const auto shape = std::vector{ cells.size() };
    cnpy::npy_save(filepath, cells.data(), shape);
}

thesis::DeviceGrid::DeviceGrid(const int3 dimensions)
    : dimensions{ dimensions } {
    const auto cell_count = dimensions.x * dimensions.y * dimensions.z;
    hipMalloc(&cells, cell_count * sizeof(FP));
}

thesis::DeviceGrid::~DeviceGrid() {
    hipFree(cells);
}

void thesis::DeviceGrid::copy(const HostGrid &grid) {
    const auto size = grid.cells.size() * sizeof(FP);
    hipMemcpy(cells, grid.cells.data(), size, hipMemcpyHostToDevice);
}
