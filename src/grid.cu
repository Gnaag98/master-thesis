#include "hip/hip_runtime.h"
#include "grid.cuh"

#include <fstream>

thesis::HostGrid::HostGrid(const int3 dimensions)
    : cells(dimensions.x * dimensions.y * dimensions.z),
      dimensions{ dimensions } {}

void thesis::HostGrid::copy(const DeviceGrid &grid) {
    const auto size = cells.size() * sizeof(float);
    hipMemcpy(cells.data(), grid.cells, size, hipMemcpyDeviceToHost);
}

void thesis::HostGrid::save(std::filesystem::path filepath) {
    auto file = std::ofstream{ filepath };
    for (const auto cell : cells) { file << std::setprecision(15) << cell << ','; }
    file << '\n';
}

thesis::DeviceGrid::DeviceGrid(const int3 dimensions)
    : dimensions{ dimensions } {
    const auto cell_count = dimensions.x * dimensions.y * dimensions.z;
    hipMalloc(&cells, cell_count * sizeof(float));
}

thesis::DeviceGrid::~DeviceGrid() {
    hipFree(cells);
}

void thesis::DeviceGrid::copy(const HostGrid &grid) {
    const auto size = grid.cells.size() * sizeof(float);
    hipMemcpy(cells, grid.cells.data(), size, hipMemcpyHostToDevice);
}
