#include "hip/hip_runtime.h"
#include "charge_density_global_2d.cuh"

__global__
void thesis::global_2d::charge_density(
    const FP *pos_x, const FP *pos_y, const size_t particle_count,
    const int3 grid_dimensions, const int cell_size, FP *densities
) {
    // Grid-stride loop. Equivalent to regular if-statement if grid is large
    // enough to cover all iterations of the loop.
    for (
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        index < particle_count;
        index += blockDim.x * gridDim.x
    ) {
        const auto position = FP3{ pos_x[index], pos_y[index], 0 };
        const auto [ u, v, w ] = cell_coordinates(position, cell_size);

        // 2D index, or center of surrounding cell closest to the origin.
        const auto i = static_cast<int>(u);
        const auto j = static_cast<int>(v);

        // Centers of all surrounding cells, named relative the indices
        // (i,j,k) of the surrounding cell closest to the origin (cell_000).
        const auto cell_000_center = int3{ i,     j    , 0 };
        const auto cell_100_center = int3{ i + 1, j    , 0 };
        const auto cell_010_center = int3{ i,     j + 1, 0 };
        const auto cell_110_center = int3{ i + 1, j + 1, 0 };

        // uvw-position relative to cell_000.
        const auto pos_rel_cell = FP3{
            u - cell_000_center.x,
            v - cell_000_center.y,
            w - cell_000_center.z
        };
        // Cell weights based on the distance to the particle.
        const auto cell_000_weight = (1 - pos_rel_cell.x) * (1 - pos_rel_cell.y);
        const auto cell_100_weight =      pos_rel_cell.x  * (1 - pos_rel_cell.y);
        const auto cell_010_weight = (1 - pos_rel_cell.x) *      pos_rel_cell.y;
        const auto cell_110_weight =      pos_rel_cell.x  *      pos_rel_cell.y;

        // Linear cell indices.
        const auto cell_000_index = cell_index(cell_000_center, grid_dimensions);
        const auto cell_100_index = cell_index(cell_100_center, grid_dimensions);
        const auto cell_010_index = cell_index(cell_010_center, grid_dimensions);
        const auto cell_110_index = cell_index(cell_110_center, grid_dimensions);

        // Weighted sum of the particle's charge.
        atomicAdd(&densities[cell_000_index], cell_000_weight);
        atomicAdd(&densities[cell_100_index], cell_100_weight);
        atomicAdd(&densities[cell_010_index], cell_010_weight);
        atomicAdd(&densities[cell_110_index], cell_110_weight);
    }
}
