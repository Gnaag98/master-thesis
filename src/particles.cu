#include "particles.cuh"

#include <fstream>

amitis::HostParticles::HostParticles(const int count, const float charge)
: pos_x(count), pos_y(count), pos_z(count), charge{ charge } {}

void amitis::HostParticles::copy(const DeviceParticles &particles) {
    const auto size = pos_x.size() * sizeof(float);
    hipMemcpy(pos_x.data(), particles.pos_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(pos_y.data(), particles.pos_y, size, hipMemcpyDeviceToHost);
    hipMemcpy(pos_z.data(), particles.pos_z, size, hipMemcpyDeviceToHost);
}

void amitis::HostParticles::save_positions(std::filesystem::path filepath) {
    auto file = std::ofstream{ filepath };
    for (const auto x : pos_x) { file << x << ','; }
    file << '\n';
    for (const auto y : pos_y) { file << y << ','; }
    file << '\n';
    for (const auto z : pos_z) { file << z << ','; }
    file << '\n';
}

amitis::DeviceParticles::DeviceParticles(const HostParticles &particles) {
    hipMalloc(&pos_x, particles.pos_x.size() * sizeof(float));
    hipMalloc(&pos_y, particles.pos_y.size() * sizeof(float));
    hipMalloc(&pos_z, particles.pos_z.size() * sizeof(float));
}

amitis::DeviceParticles::~DeviceParticles() {
    hipFree(pos_x);
    hipFree(pos_y);
    hipFree(pos_z);
}

void amitis::DeviceParticles::copy(const HostParticles &particles) {
    const auto size = particles.pos_x.size() * sizeof(float);
    hipMemcpy(pos_x, particles.pos_x.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(pos_y, particles.pos_y.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(pos_z, particles.pos_z.data(), size, hipMemcpyHostToDevice);
}
