#include "particles.cuh"

#include <fstream>

thesis::HostParticles::HostParticles(const int count, const float charge)
: pos_x(count), pos_y(count), pos_z(count), charge{ charge } {}

void thesis::HostParticles::copy(const DeviceParticles &particles) {
    const auto size = pos_x.size() * sizeof(float);
    hipMemcpy(pos_x.data(), particles.pos_x, size, hipMemcpyDeviceToHost);
    hipMemcpy(pos_y.data(), particles.pos_y, size, hipMemcpyDeviceToHost);
    hipMemcpy(pos_z.data(), particles.pos_z, size, hipMemcpyDeviceToHost);
}

void thesis::HostParticles::save_positions(std::filesystem::path filepath) {
    auto file = std::ofstream{ filepath };
    for (const auto x : pos_x) { file << x << ','; }
    file << '\n';
    for (const auto y : pos_y) { file << y << ','; }
    file << '\n';
    for (const auto z : pos_z) { file << z << ','; }
    file << '\n';
}

thesis::DeviceParticles::DeviceParticles(const HostParticles &particles) {
    const auto size = particles.pos_x.size();
    hipMalloc(&pos_x, size * sizeof(float));
    hipMalloc(&pos_y, size * sizeof(float));
    hipMalloc(&pos_z, size * sizeof(float));
}

thesis::DeviceParticles::~DeviceParticles() {
    hipFree(pos_x);
    hipFree(pos_y);
    hipFree(pos_z);
}

void thesis::DeviceParticles::copy(const HostParticles &particles) {
    const auto size = particles.pos_x.size() * sizeof(float);
    hipMemcpy(pos_x, particles.pos_x.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(pos_y, particles.pos_y.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(pos_z, particles.pos_z.data(), size, hipMemcpyHostToDevice);
}
