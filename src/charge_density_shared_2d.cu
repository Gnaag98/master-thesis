#include "hip/hip_runtime.h"
#include "charge_density_shared_2d.cuh"

#include <cub/device/device_radix_sort.cuh>

#include "common.cuh"

__global__
void thesis::shared_2d::initialize_particle_indices(
    const size_t particle_count, int *indices
) {
    // Grid-stride loop. Equivalent to regular if-statement if grid is large
    // enough to cover all iterations of the loop.
    for (
        auto index = blockIdx.x * blockDim.x + threadIdx.x;
        index < particle_count;
        index += blockDim.x * gridDim.x
    ) {
        indices[index] = index;
    }
}

__global__
void thesis::shared_2d::associate_particles_with_cells(
    const float *pos_x, const float *pos_y, const size_t particle_count,
    const int3 grid_dimensions, const int cell_size, int *cell_indices
) {
    // Grid-stride loop. Equivalent to regular if-statement if grid is large
    // enough to cover all iterations of the loop.
    for (
        auto particle_index = blockIdx.x * blockDim.x + threadIdx.x;
        particle_index < particle_count;
        particle_index += blockDim.x * gridDim.x
    ) {
        // Position in world coordinates.
        const auto position = float3{
            pos_x[particle_index], pos_y[particle_index], 0
        };
        // Position in grid coordinates, with origin at the first cell center.
        const auto [ u, v, w ] = cell_coordinates(position, cell_size);
        // 2D indices of first enclosing cell, by first meaning the one with
        // lowest index, i.e., closest to the origin.
        const auto i = static_cast<int>(floor(u));
        const auto j = static_cast<int>(floor(v));
        // Store 1D index.
        cell_indices[particle_index] = i + j * grid_dimensions.x;
    }
}

void thesis::shared_2d::sort_particles_by_cell(
    void *sort_storage, size_t &sort_storage_size,
    const int *associated_cells_in, int *associated_cells_out,
    const int *particle_indices_in, int *particle_indices_out,
    const size_t particle_count
) {
    hipcub::DeviceRadixSort::SortPairs(
        sort_storage, sort_storage_size,
        associated_cells_in, associated_cells_out,
        particle_indices_in, particle_indices_out,
        particle_count
    );
}
